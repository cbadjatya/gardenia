#include "hip/hip_runtime.h"
// Copyright 2016, National University of Defense Technology
// Authors: Xuhao Chen <cxh@illinois.edu>
#define SSSP_VARIANT "topo_opt"
#include "sssp.h"
#include "timer.h"
#include "cuda_launch_config.hpp"
#include "cutil_subset.h"



// OPT KERNELS BEGIN

__global__ void preprocess1(int N, int *P, bool *isBad, uint64_t *CSR_N, int *numBadWarps, int magic)
{
    // Set the warp reference array P. count Bad Warps and mark each bad warp in the array isBad.
    // N here is the largest multiple of 32 <= actual N.
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < N)
    {
        int wid = tid / 32;
        if (tid % 32 == 0)
            P[wid] = wid; // set the initial reference
        int max_value = (CSR_N[tid + 1] - CSR_N[tid]);
        int min_value = max_value;
        for (int i = 16; i > 0; i = i / 2)
        {
            max_value = max(max_value, __shfl_down_sync(-1, max_value, i));
            min_value = min(min_value, __shfl_down_sync(-1, min_value, i));
        }
        if (tid % 32 == 0 && max_value - min_value > magic) // value based on heuristics!
        {
            isBad[wid] = true;
            atomicAdd(numBadWarps, 1);
        }
    }
}

__global__ void preprocess2(bool *isBad, int badWarps, int *G, int *B, int totalWarps, int *Gi, int *Bi)
{
    // kernel launched with totalWarps number of threads

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    bool flag;
    if (tid < totalWarps)
    {
        flag = isBad[tid];
        if (tid < badWarps)
        {
            if (!flag)
            {
                int i = atomicAdd(Gi, 1);
                G[i] = tid;
            }
        }
        else if (flag)
        {
            int i = atomicAdd(Bi, 1);
            B[i] = tid;
        }
    }
    // launch a dynamic kernel here?
    
}

__global__ void preprocess2_1(int *P, int Gi, int *G, int *B)
{
    // kernel launched with numBadWarps number of threads.

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < Gi)
    {
        int temp1 = G[tid];
        int temp2 = B[tid];
        P[temp1] = temp2;
        P[temp2] = temp1;
    }
}

__global__ void preprocess3(int N, int numBadWarps, uint64_t* d_offset, int* thread_map, int* P, int totalWarps){

   __shared__ int ind[512][2];

    unsigned int i, ij, v, wid_orig, wid_new;

    i = threadIdx.x;
    v = i + blockIdx.x * blockDim.x;
    int id = v;

    if (v >= N) return;

    thread_map[v] = v;

    wid_orig = v / 32;
    wid_new = wid_orig;


    if (wid_new < totalWarps){
        wid_new = P[wid_new];
        v = wid_new * 32 + i % 32; // new id according to new warp arrangement
        thread_map[id] = v;
    }

    if ((blockIdx.x * blockDim.x) / 32 < numBadWarps)
    {

        ind[threadIdx.x][0] = (d_offset[v + 1] - d_offset[v]); // the loop's limit
        ind[threadIdx.x][1] = v;

        // if(i == 0) atomicAdd(oblocks,1); // DEBUG STUFF

        for (int k = 2; k <= 512; k <<= 1)
        {
            for (int j = k >> 1; j > 0; j = j >> 1)
            {

                ij = i ^ j;

                if (ij > i)
                {
                    int temp[2];
                    if (((i & k) == 0 && ind[i][0] > ind[ij][0]) || ((i & k) != 0 && ind[i][0] < ind[ij][0]))
                    {
                        temp[0] = ind[i][0];
                        temp[1] = ind[i][1];
                        ind[i][0] = ind[ij][0];
                        ind[i][1] = ind[ij][1];
                        ind[ij][0] = temp[0];
                        ind[ij][1] = temp[1];
                    }
                }
                __syncthreads();
            }
        }

        thread_map[id] = ind[threadIdx.x][1];
    }


}

// OPT KERNELS END

//Naive CUDA implementation of the Bellman-Ford algorithm for SSSP
__global__ void initialize(int m, int source, bool *visited, bool *expanded) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		expanded[id] = false;
		if(id == source) visited[id] = true;
		else visited[id] = false;
	}
}

/**
 * @brief naive Bellman_Ford SSSP kernel entry point.
 *
 * @param[in] m                 Number of vertices
 * @param[in] d_row_offsets     Device pointer of VertexId to the row offsets queue
 * @param[in] d_column_indices  Device pointer of VertexId to the column indices queue
 * @param[in] d_weight          Device pointer of DistT to the edge weight queue
 * @param[out]d_dist            Device pointer of DistT to the distance queue
 * @param[in] d_in_queue        Device pointer of VertexId to the incoming frontier queue
 * @param[out]d_out_queue       Device pointer of VertexId to the outgoing frontier queue
 */
__global__ void bellman_ford(int m, uint64_t *row_offsets, VertexId *column_indices, DistT *weight, DistT *dist, bool *changed, bool *visited, bool *expanded, int* tmap) {
	int src = blockIdx.x * blockDim.x + threadIdx.x;
    int id = src;
	if(src >= m) return;
    
    // preventing unnecessary access?? Doesn't seem like it...
    // if(blockIdx.x < numBadBlocks || isBad[src/32])
    src = tmap[src];

	if(visited[src] && !expanded[src]) { // visited but not expanded
		expanded[src] = true;
		//atomicAdd(num_frontier, 1);
		int row_begin = row_offsets[src];
		int row_end = row_offsets[src + 1];
        int dsrc = dist[src];
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = column_indices[offset];
			DistT old_dist = dist[dst]; 
			DistT new_dist = dsrc + weight[offset];
			if (new_dist < old_dist) {
				if (atomicMin(&dist[dst], new_dist) > new_dist) {
					if(expanded[dst]) expanded[dst] = false;
					*changed = true;
				}
			}
		}
	}
}

__global__ void update(int m, DistT *dist, bool *visited) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < m) {
		if(dist[id] < MYINFINITY && !visited[id])
			visited[id] = true;
	}
}

/**
 * @brief naive topology-driven mapping GPU SSSP entry point.
 *
 * @param[in] m                 Number of vertices
 * @param[in] h_row_offsets     Host pointer of VertexId to the row offsets queue
 * @param[in] h_column_indices  Host pointer of VertexId to the column indices queue
 * @param[in] h_weight          Host pointer of DistT to the edge weight queue
 * @param[out]h_dist            Host pointer of DistT to the distance queue
 */
void SSSPSolver(Graph &g, int source, DistT *h_weight, DistT *h_dist, int delta, int magic) {
	auto m = g.V();
	auto nnz = g.E();
	auto h_row_offsets = g.out_rowptr();
	auto h_column_indices = g.out_colidx();	
	//print_device_info(0);
	uint64_t *d_row_offsets;
	VertexId *d_column_indices;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(uint64_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(VertexId)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(uint64_t), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(VertexId), hipMemcpyHostToDevice));

	DistT zero = 0;
	int one = 1;
	DistT *d_weight;
	DistT * d_dist;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_weight, nnz * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_weight, h_weight, nnz * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, m * sizeof(DistT)));
	CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, m * sizeof(DistT), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(zero), hipMemcpyHostToDevice));
	
	bool *d_changed, h_changed, *d_visited, *d_expanded;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_changed, sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_visited, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_expanded, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemset(d_visited, 0, m * sizeof(bool)));
	CUDA_SAFE_CALL(hipMemcpy(&d_visited[source], &one, sizeof(bool), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemset(d_expanded, 0, m * sizeof(bool)));

	CUDA_SAFE_CALL(hipDeviceSynchronize());

	// printf("Source node neighbors : %d\n", h_row_offsets[source+1] - h_row_offsets[source]);

	
	int iter = 0;
	//int h_num_frontier = 1;
	int nthreads = 512;
	int nblocks = (m - 1) / nthreads + 1;
	printf("Launching CUDA SSSP solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);


	// Timer t1;
	// t1.Start();
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);
    // Adding OPT Code

    int *P, *d_P, numBadWarps, *d_numBadWarps;
    bool *isBad, *d_isBad;
    int totalWarps = (m / 32);
    P = (int *)malloc(totalWarps * sizeof(int));
    isBad = (bool *)malloc(totalWarps);

    numBadWarps = 0;
    hipMalloc(&d_numBadWarps, sizeof(int));
    hipMemset(d_numBadWarps, 0, sizeof(int));

    hipMalloc(&d_P, totalWarps * sizeof(int));

    hipMalloc(&d_isBad, totalWarps);
    hipMemset(d_isBad, 0, totalWarps);

    preprocess1<<<ceil(((float)totalWarps * 32) / 512), 512>>>(totalWarps * 32, d_P, d_isBad, d_row_offsets, d_numBadWarps, magic);
    // CHECK_DEBUG(hipMemcpy(P,d_P,totalWarps*sizeof(int),hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(&numBadWarps, d_numBadWarps, sizeof(int), hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(isBad,d_isBad,totalWarps,hipMemcpyDeviceToHost));

    int *d_Good, *d_Bad, *d_Gi, *d_Bi;
    int Gi, Bi;

    hipMalloc(&d_Good, numBadWarps * sizeof(int));
    hipMalloc(&d_Bad, numBadWarps * sizeof(int));
    hipMalloc(&d_Gi, sizeof(int));
    hipMalloc(&d_Bi, sizeof(int));
    hipMemset(d_Gi, 0, sizeof(int));
    hipMemset(d_Bi, 0, sizeof(int));

    preprocess2<<<ceil(totalWarps * 1.0 / 512), 512>>>(d_isBad, numBadWarps, d_Good, d_Bad, totalWarps, d_Gi, d_Bi);
    
    CUDA_SAFE_CALL(hipMemcpy(&Gi, d_Gi, sizeof(int), hipMemcpyDeviceToHost));

    //int num_bad_blocks = ceil(numBadWarps*32*1.0/nthreads);
    
    // CHECK(hipMemcpy(&Bi, d_Bi,sizeof(int), hipMemcpyDeviceToHost));

    // hipFree(d_isBad);

    preprocess2_1<<<max(1, (int)ceil(Gi * 1.0 / 512)), 512>>>(d_P, Gi, d_Good, d_Bad);

    // hipFree(d_Good);
    // hipFree(d_Bad);

    // CHECK(hipMemcpy(P,d_P,totalWarps*sizeof(int),hipMemcpyDeviceToHost));
    int* thread_mappings;
    hipMalloc(&thread_mappings, m*sizeof(int));
    preprocess3<<<nblocks, nthreads>>>(m, numBadWarps, d_row_offsets, thread_mappings, d_P, totalWarps);
    // hipDeviceSynchronize();
    // OPT code ends
    // t1.Stop();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);

    printf("time taken for opt = %f\n",elapsed);
    printf("Number of Bad Warps found = %d\n", numBadWarps);

	// Timer t;
	// t.Start();

    hipEventRecord(start, 0);


	do {
		++ iter;
		h_changed = false;
		CUDA_SAFE_CALL(hipMemcpy(d_changed, &h_changed, sizeof(bool), hipMemcpyHostToDevice));
		//CUDA_SAFE_CALL(hipMemcpy(d_num_frontier, &zero, sizeof(int), hipMemcpyHostToDevice));
		// hipProfilerStart();
		bellman_ford<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_weight, d_dist, d_changed, d_visited, d_expanded, thread_mappings);
		// hipProfilerStop();
		update<<<nblocks, nthreads>>>(m, d_dist, d_visited);
		// CudaTest("solving failed");
		CUDA_SAFE_CALL(hipMemcpy(&h_changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost));
		//CUDA_SAFE_CALL(hipMemcpy(&h_num_frontier, d_num_frontier, sizeof(int), hipMemcpyDeviceToHost));
		//printf("iteration %d: num_frontier = %d\n", iter, h_num_frontier);
	} while (h_changed);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	// t.Stop();

	hipEventRecord(stop);
    hipEventSynchronize(stop);
    elapsed = 0;
    hipEventElapsedTime(&elapsed, start, stop);

	printf("\titerations = %d.\n", iter);
	printf("\truntime [%s] = %f ms.\n", SSSP_VARIANT, elapsed);

	CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, m * sizeof(DistT), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_weight));
	CUDA_SAFE_CALL(hipFree(d_dist));
	CUDA_SAFE_CALL(hipFree(d_changed));
	// CUDA_SAFE_CALL(hipFree(d_num_frontier));
	return;
}

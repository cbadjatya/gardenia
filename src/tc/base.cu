#include "hip/hip_runtime.h"
// Copyright (c) 2016, Xuhao Chen
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <vector>
#include <algorithm>
#include "tc.h"
#include "timer.h"
#include "cutil_subset.h"
#include "cuda_launch_config.hpp"

__global__ void ordered_count(int m, IndexT *row_offsets, IndexT *column_indices, int *total) {
	typedef hipcub::BlockReduce<int, BLOCK_SIZE> BlockReduce;
	__shared__ typename BlockReduce::TempStorage temp_storage;
	int u = blockIdx.x * blockDim.x + threadIdx.x;
	int local_total = 0;
	if (u < m) {
		auto begin_u = row_offsets[u];
		auto end_u = row_offsets[u+1]; 
		for (auto off_u = begin_u; off_u < end_u; ++ off_u) {
			int v = column_indices[off_u];
			int begin_v = row_offsets[v];
			int end_v = row_offsets[v+1];
      auto it = begin_u;
			for (auto off_v = begin_v; off_v < end_v; ++ off_v) {
				int w = column_indices[off_v];
				while (column_indices[it] < w && it < end_u) it ++;
				if (it != end_u && column_indices[it] == w) local_total += 1;
			}
		}
	}
	int block_total = BlockReduce(temp_storage).Sum(local_total);
	if(threadIdx.x == 0) atomicAdd(total, block_total);
}

void TCSolver(Graph &g, uint64_t &total) {
	int64_t m = g.num_vertices();
	int64_t nnz = g.num_edges();
	IndexT *h_row_offsets = g.out_rowptr();
	IndexT *h_column_indices = g.out_colidx();
	//print_device_info(0);
	int zero = 0;
	int *d_row_offsets, *d_column_indices;//, *d_degree;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_row_offsets, (m + 1) * sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_column_indices, nnz * sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_row_offsets, h_row_offsets, (m + 1) * sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(d_column_indices, h_column_indices, nnz * sizeof(int), hipMemcpyHostToDevice));
	int h_total = 0, *d_total;
	CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpy(d_total, &zero, sizeof(int), hipMemcpyHostToDevice));

	int nthreads = BLOCK_SIZE;
	int nblocks = (m - 1) / nthreads + 1;
	int max_blocks = maximum_residency(ordered_count, nthreads, 0);
	printf("Launching CUDA TC solver (%d CTAs, %d threads/CTA) ...\n", nblocks, nthreads);

	Timer t;
	t.Start();
	ordered_count<<<nblocks, nthreads>>>(m, d_row_offsets, d_column_indices, d_total);
	CudaTest("solving failed");
	CUDA_SAFE_CALL(hipDeviceSynchronize());
	t.Stop();

	printf("\truntime [cuda_base] = %f ms.\n", t.Millisecs());
	CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(int), hipMemcpyDeviceToHost));
	total = (uint64_t)h_total;
	CUDA_SAFE_CALL(hipFree(d_row_offsets));
	CUDA_SAFE_CALL(hipFree(d_column_indices));
	CUDA_SAFE_CALL(hipFree(d_total));
}

